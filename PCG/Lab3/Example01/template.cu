#include "hip/hip_runtime.h"
/**
 * @file      template.cpp
 *
 * @author    Jiri Jaros \n
 *            Faculty of Information Technology \n
 *            Brno University of Technology \n
 *            jarosjir@fit.vutbr.cz
 *
 * @brief     PC lab 3 / Texture memory with data in global memory
 *
 * @details   This sample takes an input PGM image (image_filename) and generates an output PGM image
 *            (image_filename_out).  This CUDA kernel performs a simple 2D transform (rotation) on the texture
 *            coordinates (u,v).
 *
 * @version   2022
 *
 * @date      27 October   2020, 11:36 (created) \n
 * @date      27 October   2022, 10:02 (revised by Ondrej Olsak)
 *
 */

//--------------------------------------------------------------------------------------------------------------------//
// Useful links:                                                                                                      //
//   i.   CUDA SDK documentation: https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html                    //
//   ii.  List of CUDA routines:  https://docs.nvidia.com/cuda/cuda-runtime-api/index.html                            //
//   iii. Texture memory:         https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#texture-memory     //
//--------------------------------------------------------------------------------------------------------------------//

// Includes, system
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// Includes CUDA
#include <hip/hip_runtime.h>
// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check


/// Define the files that are to be save and the reference images for validation
const char* imageFilename = "lena.pgm";
const char* refFilename   = "lena_ref.pgm";

/// Max acceptable error
constexpr float makEpsilonError = 5e-3f;
// angle to rotate image by (in radians)
constexpr float rotationAngle   = 0.5f;


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//                                       Functions to be implemented                                                  //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

/**
 * Transform an image using texture lookups
 * @param [out] outputImage - Output image
 * @param [in]  texImage    - Cuda texture object
 * @param [in]  height      - Image height
 * @param [in]  width       - Image width
 * @param [in]  float       - Rotation angle
 */
__global__ void transformKernel(float*      outputImage,
                                hipTextureObject_t texImage,
                                const int   height,
                                const int   width,
                                const float theta)
{
  // Get position of the thread in the image
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  // Read from texture and write to global memory
  if (y < height && x < width)
  {
    // Normalized coordinates <0,1>
    float u = x / float(width);
    float v = y / float(height);

    // Move coordinates to the middle of the image
    u -= 0.5f;
    v -= 0.5f;

    // Calculate rotated coordinates
    const float tu = u * cosf(theta) - v * sinf(theta) + 0.5f;
    const float tv = v * cosf(theta) + u * sinf(theta) + 0.5f;

    //----------------------------------------------------------------------------------------------------------------//
    // 1. Write pixel [tu, tv] into outputImage                                                                       //
    //----------------------------------------------------------------------------------------------------------------//


    //----------------------------------------------------------------------------------------------------------------//
  }
} // end of transformKernel
//----------------------------------------------------------------------------------------------------------------------

/**
 * Run a simple test for CUDA
 */
bool runTest(int argc, char** argv)
{
  char* imagePath = sdkFindFilePath(imageFilename, argv[0]);

  if (imagePath == nullptr)
  {
    printf("Unable to source image file: %s\n", imageFilename);
    exit(EXIT_FAILURE);
  }

  // Image size
  unsigned int width  = 0;
  unsigned int height = 0;
  // load image from disk
  float* hImage = nullptr;

  sdkLoadPGM(imagePath, &hImage, &width, &height);

  const int size = width * height;
  printf("Loaded '%s', %d x %d pixels\n", imageFilename, width, height);

  //  Load reference image from image (output)
  float* hRefImage    = (float *) malloc(size * sizeof(float));
  // Allocate mem for the result on host side
  float* hOutputImage = (float *) malloc(size * sizeof(float));

  char*  refPath      = sdkFindFilePath(refFilename, argv[0]);

  if (refPath == nullptr)
  {
    printf("Unable to find reference image file: %s\n", refFilename);
    exit(EXIT_FAILURE);
  }

  sdkLoadPGM(refPath, &hRefImage, &width, &height);


  // Time measurement
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  //                                              Tasks to be implemented                                             //
  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  size_t pitch;

  float* dOutputImage = nullptr;
  float* dInputImage  = nullptr;

  // Allocate GPU memory for input and output image here
  hipMallocPitch<float>(&dInputImage, &pitch, width*sizeof(float), height);
  hipMalloc<float>(&dOutputImage, size * sizeof(float));

  // Copy input image on the GPU
  hipMemcpy2D(dInputImage, pitch, hImage, width*sizeof(float), width*sizeof(float),
               height, hipMemcpyHostToDevice);

  //------------------------------------------------------------------------------------------------------------------//
  // 2. Create Channel Format                                                                                         //
  //------------------------------------------------------------------------------------------------------------------//

  // Create Texture descriptor
  hipChannelFormatDesc channelDesc;
  // use cudaCrateChannelDesc
  // Lecture no.5, slide 30
  channelDesc = ;

  //------------------------------------------------------------------------------------------------------------------//
  // 4. Set texture parameters                                                                                        //
  //    https://docs.nvidia.com/cuda/cuda-runtime-api/structcudaTextureDesc.html#structcudaTextureDesc
  //------------------------------------------------------------------------------------------------------------------//
  struct hipTextureDesc textureDesc;
  memset(&textureDesc, 0, sizeof(textureDesc));

  // hipAddressModeWrap        Wrapping address mode
  // hipAddressModeClamp       Clamp to edge address mode
  // hipAddressModeMirror      Mirror address mode
  // hipAddressModeBorder      Border address mode
  textureDesc.addressMode[0] = ; // ?
  textureDesc.addressMode[1] = ; // ?

  // hipFilterModePoint  Point filter mode
  // hipFilterModeLinear Linear filter mode
  textureDesc.filterMode = ; //?

  // access with normalized texture coordinates
  textureDesc.normalizedCoords = ; //?

  //------------------------------------------------------------------------------------------------------------------//
  // 5. Set resource descriptor
  //    https://docs.nvidia.com/cuda/cuda-runtime-api/structcudaResourceDesc.html#structcudaResourceDesc
  //------------------------------------------------------------------------------------------------------------------//
  struct hipResourceDesc resourceDesc;
  memset(&resourceDesc, 0, sizeof(resourceDesc));

  resourceDesc.resType = ;
  resourceDesc.res.pitch2D.devPtr = ;
  resourceDesc.res.pitch2D.desc = ;
  resourceDesc.res.pitch2D.width  = ;
  resourceDesc.res.pitch2D.height = ;
  resourceDesc.res.pitch2D.pitchInBytes = ;

  //------------------------------------------------------------------------------------------------------------------//
  // 6. Create texture object                                                                                //
  //------------------------------------------------------------------------------------------------------------------//

  hipTextureObject_t texImage;

  //------------------------------------------------------------------------------------------------------------------//

  // Initialize the grid and block dimensions
  dim3 dimBlock(16, 16);
  dim3 dimGrid(width / dimBlock.x, height / dimBlock.y);

  // Launch the GPU kernel
  hipEventRecord(start, 0);
  // Execute the kernel
  transformKernel<<<dimGrid, dimBlock, 0>>>(dOutputImage, texImage, height, width, rotationAngle);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  float elapsedTime = 0;
  hipEventElapsedTime(&elapsedTime, start, stop);

  printf("Processing time:     %f (ms)\n", elapsedTime);
  printf("Processing perf: %.2f Mpixels/sec\n", (width * height / (elapsedTime / 1000.0f)) / 1e6);

  // Copy result from device to host
  hipMemcpy(hOutputImage, dOutputImage, size * sizeof(float), hipMemcpyDeviceToHost);

  //------------------------------------------------------------------------------------------------------------------//
  // 7. Unbind texture and free memory                                                                               //
  //------------------------------------------------------------------------------------------------------------------//


  //------------------------------------------------------------------------------------------------------------------//

  //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

  hipFree(dOutputImage);
  hipFree(dInputImage);

  // Write result to file
  char outputFilename[1024];
  strcpy(outputFilename, imagePath);
  strcpy(outputFilename + strlen(imagePath) - 4, "_out.pgm");
  sdkSavePGM(outputFilename, hOutputImage, width, height);
  printf("Wrote '%s'\n", outputFilename);

  bool testResult = false;
  // Write regression file if necessary
  if (checkCmdLineFlag(argc, (const char **) argv, "regression"))
  {
    // Write file for regression test
    sdkWriteFile<float>("./data/regression.dat", hOutputImage, width * height,  0.0f, false);
  }
  else
  {
    // We need to reload the data from disk,
    // because it is inverted upon output
    sdkLoadPGM(outputFilename, &hOutputImage, &width, &height);

    printf("Comparing files\n");
    printf("\toutput:    <%s>\n", outputFilename);
    printf("\treference: <%s>\n", refPath);

    testResult = compareData(hOutputImage, hRefImage, width * height, makEpsilonError, 0.15f);
  }

  hipEventDestroy(start);
  hipEventDestroy(stop);

  free(imagePath);
  free(refPath);
  return testResult;
}// end of runTest
//----------------------------------------------------------------------------------------------------------------------
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

/**
 *  Main routine
 * @param argc
 * @param argv
 * @return
 */
int main(int argc, char** argv)
{
  // Process command-line arguments
  if (argc > 1)
  {
    if (checkCmdLineFlag(argc, (const char **) argv, "input"))
    {
      getCmdLineArgumentString(argc, (const char **) argv, "input", (char **) &imageFilename);

      if (checkCmdLineFlag(argc, (const char **) argv, "reference"))
      {
        getCmdLineArgumentString(argc, (const char **) argv, "reference", (char **) &refFilename);
      }
      else
      {
        printf("-input flag should be used with -reference flag\n");
        exit(EXIT_FAILURE);
      }
    }
    else if (checkCmdLineFlag(argc, (const char **) argv, "reference"))
    {
      printf("-reference flag should be used with -input flag\n");
      exit(EXIT_FAILURE);
    }
  }

  const bool testResult = runTest(argc, argv);

  printf("Completed, returned %s\n", testResult ? "OK" : "ERROR!");
  exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}// end of main
//----------------------------------------------------------------------------------------------------------------------
