#include "hip/hip_runtime.h"
/**
 * @file      template.cpp
 *
 * @author    Jiri Jaros \n
 *            Faculty of Information Technology \n
 *            Brno University of Technology \n
 *            jarosjir@fit.vutbr.cz
 *
 * @brief     PC lab 4 / Reduction
 *
 * @version   2021
 *
 * @date      06 November  2020, 10:02 (created) \n
 * @date      11 November  2020, 10:03 (revised) \n
 *
 */
#include <wb.h>

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//                                       Function to be implemented                                                   //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

/**
 * Calculate the sum of the vector elements.
 * @param [in] output - Output value
 * @param [in] input  - Input array
 * @param [in] size   - Size of the input array
 */
__global__ void cudaReduction(int*       output,
                              const int* input,
                              const int  size)
{
  // Load a segment of the input vector into shared memory
  extern __shared__ int partialSum[];


  // Clear partial sum
  partialSum[threadIdx.x] = 0;
  __syncthreads();

  // Reduce the array into partial sums
  int threadId = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = threadId; i < size; i += gridDim.x * blockDim.x) {
    partialSum[threadIdx.x] += input[i]; 
  }
  __syncthreads();


  // Reduction kernel
  // Traverse the reduction tree.
  for (int stride = blockDim.x >> 1; stride >= 1; stride >>= 1) {
    if (threadIdx.x < stride) {
      partialSum[threadIdx.x] += partialSum[threadIdx.x + stride];
    }
    __syncthreads();
  }



  // Write the computed sum of the block to the output vector at the
  if (threadIdx.x == 0) {
    atomicAdd(output, partialSum[0]);
  }


}// end of cudaReduction
//----------------------------------------------------------------------------------------------------------------------

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

/**
 * Main routine - complete necessary tasks.
 * @param argc
 * @param argv
 * @return
 */
int main(int argc, char **argv)
{
  // Number of elements in the input list
  int inputLength;

  // Parse command line.
  wbArg_t args = wbArg_read(argc, argv);

  int* hostInput  = (int *) wbImport(wbArg_getInputFile(args, 0), &inputLength, "Integer");
  // Only one element
  int* hostOutput = (int *) malloc(sizeof(int));

  printf("----------------- Example 3: CUDA reduction in shared memory -----------------\n");
  printf("Input size: %d\n", inputLength);

  printf("Input sequence: ");
  for (int i = 0 ; i < inputLength ; i++)
  {
    printf("%d, ",hostInput[i]);
  }
  printf("\n");

  int* deviceInput;
  int* deviceOutput;
  hipMalloc<int>(&deviceInput,  sizeof(int) * inputLength);
  hipMalloc<int>(&deviceOutput, sizeof(int));


  //Copy memory to the GPU
  hipMemcpy(deviceInput, hostInput, sizeof(int) * inputLength, hipMemcpyHostToDevice);
  hipMemset(deviceOutput, sizeof(int), 0);

  // Initialize the grid and block dimensions here
  dim3 dimGrid(64, 1, 1);
  dim3 dimBlock(256, 1, 1);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);

  // Launch the GPU Kernel here
  cudaReduction<<<dimGrid, dimBlock, 256 * sizeof(int)>>>(deviceOutput, deviceInput, inputLength);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  float time = 0;
  hipEventElapsedTime(&time, start, stop);

  printf("Time to reduce in shared memory: %4.2f ms\n", time);

  // Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, deviceOutput, sizeof(int), hipMemcpyDeviceToHost);
  printf("Final sum: %d\n", hostOutput[0]);

  // Free the GPU memory here
  hipFree(deviceInput);
  hipFree(deviceOutput);

  float sol = float(hostOutput[0]);
  wbSolution(args, &sol, 1);

  free(hostInput);
  free(hostOutput);

  return 0;
}// end of main
//----------------------------------------------------------------------------------------------------------------------
