#include "hip/hip_runtime.h"
/**
 * @file      solution.cpp
 *
 * @author    Jiri Jaros \n
 *            Faculty of Information Technology \n
 *            Brno University of Technology \n
 *            jarosjir@fit.vutbr.cz
 *
 * @brief     PC lab 4 / Histogram
 *
 * @version   2021
 *
 * @date      06 November  2020, 10:02 (created) \n
 * @date      06 November  2020, 13:06 (revised) \n
 *
 */

#include <wb.h>

#include <string.h>

using std::string;

//--------------------------------------------------------------------------------------------------------------------//
// Useful links:                                                                                                      //
//   i. CUDA SDK documentation https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html                       //
//   ii. List of CUDA routines https://docs.nvidia.com/cuda/cuda-runtime-api/index.html                               //
//--------------------------------------------------------------------------------------------------------------------//

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//                                       Function to be implemented                                                   //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

//--------------------------------------------------------------------------------------------------------------------//
// 1. Implement the CUDA kernel for calculating histogram in global memory                                            //
//--------------------------------------------------------------------------------------------------------------------//

/**
 * CUDA kernel to compute histogram in global memory.
 * @param [out] histogram   - Histogram
 * @param [in]  input       - Input text
 * @param [in]  numBins     - Number of bins
 * @param [in]  inputLenght - Size of the input text
 */
 __global__ void cudaHistogramGlobal(unsigned int* histogram,
                                     unsigned int  numBins,
                                     const char*   input,
                                     unsigned int  inputLenght)
{
  // Thread ID and stride (grid size).



  // Traverse through the text and store the histogram into global memory.
  // - If there's no bin for a given char, use the last one.
  // - Remember, the input may be longer than the total number of threads.




}// end of cudaHistogramGlobal
//----------------------------------------------------------------------------------------------------------------------

//--------------------------------------------------------------------------------------------------------------------//
// 2. Implement the routine for calculating histogram in global memory                                                //
//--------------------------------------------------------------------------------------------------------------------//

/**
 * Compute histogram using global memory.
 * @param [out] histogram   - Histogram
 * @param [in]  input       - Input text
 * @param [in]  numBins     - Number of bins
 * @param [in]  inputLenght - Size of the input text
 */
void histogramGlobal(unsigned int* histogram,
                     unsigned int  numBins,
                     const char*   input,
                     unsigned int  inputLenght)
{
  // Zero the histogram in global memory.



  // Launch histogram kernel on the bins
  dim3 blockDim(256), gridDim(64);
  // Launch the kernel.



}// end of histogramGlobal
//----------------------------------------------------------------------------------------------------------------------

/**
 * Main routine - complete necessary tasks.
 * @param argc
 * @param argv
 * @return
 */
int main(int argc, char *argv[])
{
  // Parse command line.
  wbArg_t args = wbArg_read(argc, argv);

  // Number of bins in the histogram
  constexpr unsigned int nBins = 128;
  // Input length
  int inputLength = 0;

  // Read input files
  char*          hostInput      = (char *) wbImport(wbArg_getInputFile(args, 0), &inputLength, "Text");
  unsigned int*  hostHistogram  = (unsigned int *) malloc(nBins * sizeof(unsigned int));

  printf("-------------------------- Example 1: CUDA histogram--------------------------\n");
  printf("Input size:     %d\n", inputLength);
  printf("Histogram size: %d\n", nBins);


  // Allocate device memory for histogram and input data
  char*         deviceInput;
  unsigned int* deviceHistogram;

  hipMalloc<char>(&deviceInput, inputLength * sizeof(char));
  hipMalloc<unsigned int>(&deviceHistogram, nBins * sizeof(unsigned int));

  // Copy input data to GPU
  hipMemcpy(deviceInput, hostInput, inputLength, hipMemcpyHostToDevice);

  // Shared histogram
  hipEvent_t start, stop;

  //------------------------------------------------ Global histogram ------------------------------------------------//
  
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);
  // Launch kernel
  histogramGlobal(deviceHistogram, nBins, deviceInput, inputLength);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  float time = 0;
  hipEventElapsedTime(&time, start, stop);

  printf("Time to histogram in global memory:  %4.2f ms\n", time);

  // Copy the GPU memory back to the CPU here
  hipMemcpy(hostHistogram, deviceHistogram, nBins * sizeof(unsigned int), hipMemcpyDeviceToHost);
  wbSolution(args, hostHistogram, nBins);
   

  hipFree(deviceInput);
  hipFree(deviceHistogram);

  free(hostHistogram);
  free(hostInput);

  return 0;
}// end of main
//----------------------------------------------------------------------------------------------------------------------
